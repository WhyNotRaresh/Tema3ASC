#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

#define MAX_LOAD_FACTOR		.9f
#define MIN_LOAD_FACTOR		.85f


// Calculeaza hash-ul unei chei prin algoritmul descris aici
// https://gist.github.com/badboy/6267743
// Dintre functiile propuse de Bob Jenkins, aceasta avea
// performantele cele mai bune
static __device__ size_t computeHash(int key)
{
	size_t hash = (size_t)key;

	hash = ~hash + (hash << 15);
	hash = hash ^ (hash >> 12);
	hash = hash + (hash << 2);
	hash = hash ^ (hash >> 4);
	hash = (hash + (hash << 3)) + (hash << 11);
	hash = hash ^ (hash >> 16);

	return hash;
}

// Kernelul se ocupa cu inserarea unui singur element in hahstable folosind
// tehnica "linear probing".
static __global__ void kernel_insert(Entry* hashMap, int* devKeys,
	int* devValues, int* numUpdates, size_t capacity)
{
	int oldKey;
	bool inserted = false;
	size_t hash;
	Entry insertedEntry;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > capacity)
	{
		return;
	}

	// se calculeaza hashul initial
	hash = computeHash(devKeys[idx]) % capacity;
	insertedEntry = {devKeys[idx], devValues[idx]};

	// Se parcurg indecsii in ordine incepand de la `hash` si se cauta o pozitie
	// libera sau pe care se afla aceeasi cheie (caz de update)
	for (; !inserted; hash = (hash + 1) % capacity)
	{
		// Cheia veche se schimba cu cea noua doar daca aceasta era KEY_INVALID
		// (0).
		oldKey = atomicCAS(&hashMap[hash].key, KEY_INVALID, insertedEntry.key);

		// In situatia in care cheia era `KEY_INVALID` (locul era liber) sau era
		// aceeasi cu noua cheie (update), valoarea se modifica
		if (KEY_INVALID == oldKey || insertedEntry.key == oldKey)
		{
			if (oldKey == insertedEntry.key)
			{
				atomicAdd(numUpdates, 1);
			}

			hashMap[hash].value = insertedEntry.value;
			inserted = true;
		}
	}
}

// Kernelul cauta sa puna in vectorul `values` valoarea corespunzatoare cheii
// indexului sau
static __global__ void kernel_search(Entry* hashMap, int* devKeys, int* values,
	size_t capacity, int numKeys)
{
	bool found = false;
	size_t hash;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > numKeys)
	{
		return;
	}

	// Initial se calculeaza hashul cheii care ii revine threadului curent.
	hash = computeHash(devKeys[idx]) % capacity;

	// Exact ca la inert, se parcurg indecsii pe rand, cautandu-se cel la care
	// este stocata de fapt cheia.
	for (; !found; hash = (hash + 1) % capacity)
	{
		if (devKeys[idx] == hashMap[hash].key)
		{
			values[idx] = hashMap[hash].value;
			found = true;
		}
	}
}

// Kernelul rehashuieste cheia din bucketurile vechi care corespunde fiecarui
// thread si o plaseaza impreuna cu valoarea sa in noul set de bucketuri.
// Daca cheia la pozitia care ii corespunde, threadul nu gaseste un element,
// acesta se termina imediat. 
static __global__ void kernel_rehash(Entry* resizedHashMap, Entry* hashMap,
	size_t initialCapacity, int finalCapacity)
{
	bool reinserted = false;
	size_t hash;
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx > initialCapacity || KEY_INVALID == hashMap[idx].key)
	{
		return;
	}

	hash = computeHash(hashMap[idx].key) % finalCapacity;

	// Se urmeaza aceeasi logica de la `kernel_insert`.
	for (; !reinserted; hash = (hash + 1) % finalCapacity)
	{
		if (KEY_INVALID ==
			atomicCAS(&resizedHashMap[hash].key, KEY_INVALID, hashMap[idx].key))
		{
			resizedHashMap[hash].value = hashMap[idx].value;
			reinserted = true;
		}
	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) :
	capacity(size), size(0)
{
	hipError_t ret;
	size_t numBytes = capacity * sizeof(*hashMap);

	ret = hipMalloc(&hashMap, numBytes);
	ASSERT(ret, "hipMalloc(hashMap) failed", exit(errno));

	// O pozitie din hashtable nu contine un element daca cheia de pe aceasta
	// este 0
	ret = hipMemset(hashMap, 0, numBytes);
	ASSERT(ret, "hipMemset(hashMap) failed", exit(errno));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable()
{
	hipFree(hashMap);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape)
{
	hipError_t ret;
	int numBlocks;
	int numThreads;
	Entry* resizedHashMap;
	size_t numBytes = numBucketsReshape * sizeof(*resizedHashMap);

	ret = hipMalloc(&resizedHashMap, numBytes);
	ASSERT(ret, "hipMalloc(hashMap) failed", exit(errno));

	ret = hipMemset(resizedHashMap, 0, numBytes);
	ASSERT(ret, "hipMemset(resizedHashMap) failed", exit(EXIT_FAILURE));

	ret = getNumBlocksThreads(numBlocks, numThreads, capacity);
	ASSERT(ret, "getNumBlocksThreads() failed", exit(EXIT_FAILURE));

	kernel_rehash<<<numBlocks, numThreads>>>(resizedHashMap, hashMap,
		capacity, numBucketsReshape);

	ret = hipDeviceSynchronize();
	ASSERT(ret, "hipDeviceSynchronize() failed", exit(EXIT_FAILURE));

	ret = hipFree(hashMap);
	ASSERT(ret, "hipFree(hashMap) failed", exit(EXIT_FAILURE));

	hashMap = resizedHashMap;
	capacity = numBucketsReshape;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int* keys, int* values, int numKeys)
{
	hipError_t ret;
	int numBlocks;
	int numThreads;
	int* numUpdates;  // cateva chei vor fi update-uri
	int* devKeys;  // se vor copia cheile pe GPU
	int* devValues;  // aceeasi copiere se face si pentru valori
	size_t numBytes = numKeys * sizeof(*devKeys);

	ret = hipMalloc(&devKeys, numBytes);
	ASSERT(ret, "hipMalloc(devKeys) failed", return false);

	ret = hipMemcpy(devKeys, keys, numBytes, hipMemcpyHostToDevice);
	ASSERT(ret, "hipMemcpy(devKeys) failed", return false);

	ret = hipMalloc(&devValues, numBytes);
	ASSERT(ret, "hipMalloc(devValues) failed", return false);

	ret = hipMemcpy(devValues, values, numBytes, hipMemcpyHostToDevice);
	ASSERT(ret, "hipMemcpy(devValues) failed", return false);

	ret = hipMallocManaged(&numUpdates, sizeof(*numUpdates));
	ASSERT(ret, "hipMallocManaged(numUpdates) failed", return false);

	// Hashtable-ul isi modifica dimensiunea cand se depaseste procentajul
	// maxim de umplere.
	if ((size + numKeys) / float(capacity) >= MAX_LOAD_FACTOR)
	{
		reshape((size + numKeys) / MIN_LOAD_FACTOR);
	}

	ret = getNumBlocksThreads(numBlocks, numThreads, numKeys);
	ASSERT(ret, "getNumBlocksThreads() failed", return false);

	kernel_insert<<<numBlocks, numThreads>>>(hashMap, devKeys, devValues,
		numUpdates, capacity);

	ret = hipDeviceSynchronize();
	ASSERT(ret, "hipDeviceSynchronize() failed", return false);

	// S-au adaugat `numKeys` - cheile care au fost updatate (numUpdates).
	size += numKeys - *numUpdates;

	ret = hipFree(devKeys);
	ASSERT(ret, "hipFree(devKeys) failed", return false);

	ret = hipFree(devValues);
	ASSERT(ret, "hipFree(devValues) failed", return false);

	ret = hipFree(numUpdates);
	ASSERT(ret, "hipFree(numUpdates)", return false);

	return true;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys)
{
	hipError_t ret;
	int numBlocks;
	int numThreads;
	int* devKeys;
	int* values;
	size_t numBytes = numKeys * sizeof(*devKeys);

	ret = hipMalloc(&devKeys, numBytes);
	ASSERT(ret, "hipMalloc(devKeys) failed", return NULL);

	ret = hipMemcpy(devKeys, keys, numBytes, hipMemcpyHostToDevice);
	ASSERT(ret, "hipMemcpy(devKeys) failed", return NULL);

	ret = hipMallocManaged(&values, numBytes);
	ASSERT(ret, "hipMallocManaged(values) failed", return NULL);

	ret = getNumBlocksThreads(numBlocks, numThreads, numKeys);
	ASSERT(ret, "getNumBlocksThreads() failed", return NULL);

	kernel_search<<<numBlocks, numThreads>>>(hashMap, devKeys, values,
		capacity, numKeys);

	hipDeviceSynchronize();

	ret = hipFree(devKeys);
	ASSERT(ret, "hipMalloc(devKeys) failed",);

	return values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor()
{
	return (float)size / capacity; // no larger than 1.0f = 100%
}

hipError_t GpuHashTable::getNumBlocksThreads(int& numBlocks, int& numThreads,
	int numItems)
{
	hipError_t ret;
	hipDeviceProp_t devProp;

	// Se presupune ca toate placile sunt de acelasi tip, motiv pentru care se
	// interogheaza placa 0.
	ret = hipGetDeviceProperties(&devProp, 0);
	ASSERT(ret, "hipGetDeviceProperties failed", return ret);

	numThreads = devProp.maxThreadsPerBlock;
	numBlocks = numItems / numThreads;

	if (numBlocks * numThreads != numItems)
	{
		++numBlocks;
	}

	return hipSuccess;
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
